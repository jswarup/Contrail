#include "hip/hip_runtime.h"
 // test.cu ___________________________________________________________________________________________________________________

#include "particle.h"
#include <stdlib.h>
#include <stdio.h>

//_____________________________________________________________________________________________________________________________

__global__ void advanceParticles(float dt, Particle * pArray, int nParticles)
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if(idx < nParticles)
	{
		pArray[idx].advance(dt);
	}
}

//_____________________________________________________________________________________________________________________________


int main(int argc, char ** argv)
{

	int		n = 1000000;
	if (argc > 1) 
		n = atoi(argv[1]);      // Number of particles
	if (argc > 2) 
		srand(atoi(argv[2]));	// Random seed

	Fl_CUDAERROR_CHECK()

	Particle	*pArray = new Particle[n];
	Particle	*devPArray = NULL;
	hipMalloc( &devPArray, n * sizeof(Particle));
	hipDeviceSynchronize(); 
	Fl_CUDAERROR_CHECK()

	hipMemcpy(devPArray, pArray, n * sizeof( Particle), hipMemcpyHostToDevice);
	hipDeviceSynchronize(); 
	Fl_CUDAERROR_CHECK()

	for (int i = 0; i < 100; i++)
	{
		float dt = (float)rand() / (float)RAND_MAX; // Random distance each step
		advanceParticles << < 1 + n / 256, 256 >> > (dt, devPArray, n);
		 
		Fl_CUDAERROR_CHECK()
		hipDeviceSynchronize();
	}
	hipMemcpy(pArray, devPArray, n * sizeof(Particle), hipMemcpyDeviceToHost);

	PointF3			totalDistance(0, 0, 0);
	PointF3			temp;
	for (int i = 0; i < n; i++)
	{
		temp = pArray[i].TotalDistance();
		totalDistance.x += temp.x;
		totalDistance.y += temp.y;
		totalDistance.z += temp.z;
	}
	float		avgX = totalDistance.x / (float)n;
	float		avgY = totalDistance.y / (float)n;
	float		avgZ = totalDistance.z / (float)n;
	float		avgNorm = sqrt(avgX * avgX + avgY * avgY + avgZ * avgZ);
	printf("Moved %d particles 100 steps. Average distance traveled is |(%f, %f, %f)| = %f\n", n, avgX, avgY, avgZ, avgNorm);
	return 0;
}
